#include "hip/hip_runtime.h"
/* Grid.cu
 * ---------------
 * This program aims to calculate the average temerature of a 2D array using
 * single and parallel approaches.  The border of the 2D array will contain
 * preset temperatures which will not change. To calculate the temperature of
 * the inner elements, the average of the four surrounding elements is calculated.
 * The program shall iterate over the array until the sum of error, or difference
 * between new values being calculated and old values, is sufficiantly low.
 *
 * @author Alex Kraemer
 * @version October 28, 2014
 */

#include <stdlib.h>
#include <stdio.h>
#include "Plate.h"

 __global__ void kernel(float** plateGrid)
{

    int posX = blockIdx.x * blockDim.x + threadIdx.x;
    int posY = blockIdx.y * blockDim.y + threadIdx.y;

	temp = plateGrid[posX + 1][posY] + plateGrid[posX][posY + 1] + plateGrid[posX - 1][posY] + plateGrid [posX][posY - 1];
	err += abs((temp *.25) - plateGrid[posX][posY]);
	plateGrid[posX][posY] = (temp *.25);
}

/*
* calculateGridAverage returns the average temerature of the grid.
*/
float calculateGridAverage(float** plateGrid)
{
	float tempAverage = 0;
	for (int i = 0; i < gridSize; i++)
	{
		for (int j = 0; j < gridSize; j++)
		{
			tempAverage += plateGrid[i][j];
		}
	}
	return (tempAverage / (gridSize * gridSize));
}


/*
* solveGrid iterates over the 2D array and calculates the new average for
* for each cell by taking the average of the cells adjacent to it.  After
* a new value is calculated, the cell's error (abs new - old) is calculated
* and added to this iterations total error.  The new value is then placed
* in that cell.
*/
float solveGrid(float** plateGrid)
{
	double temp;
	double err = 0;
	for (int i = 1; i < gridSize - 1; i++)
	{
		for (int j = 1; j < gridSize - 1; j++)
		{
			temp = plateGrid[i + 1][j] + plateGrid[i][j + 1] + plateGrid[i - 1][j]
				+ plateGrid [i][j - 1];

			err += abs((temp *.25) - plateGrid[i][j]);
			plateGrid[i][j] = (temp *.25);
		}
	}
	return err;
}

/*
* Corners need only to be calculated once, as border values do not change.
*/
void solveCorners(float** plateGrid)
{
	plateGrid[0][0] = ((plateGrid[0][1] + plateGrid[1][0]) * .5);

	plateGrid[0][gridSize - 1] = ((plateGrid[0][gridSize - 2]
		+ plateGrid[1][gridSize - 1]) * .5);

	plateGrid[gridSize - 1][gridSize - 1] = ((plateGrid[gridSize - 1][gridSize - 2]
		+ plateGrid[gridSize - 2][gridSize - 1]) * .5);

	plateGrid[gridSize - 1][0] = ((plateGrid[gridSize - 1][1]
		+ plateGrid[gridSize - 2][0]) * .5);
}

/*
* makeGrid sets each edge of the 2D array to the desired values.
*/
void makeGrid(float** plateGrid)
{
	for (int i = 1; i < gridSize - 1; i++)
	{
		plateGrid[i][0] = (44.0f);
		plateGrid[i][gridSize - 1] = (80.0f);
		plateGrid[0][i] = (25.0f);
		plateGrid[gridSize - 1][i] = (92.0f);
	}
}

int main()
{

	/*
	* Single threaded solution for solving the grid.
	*/

	float** grid;

	grid = (float**)malloc(gridSize * sizeof(float*));

	for (int i = 0; i < gridSize; i++)
	{
  		grid[i] = (float*)malloc(gridSize * sizeof(float));
	}

	makeGrid(grid);
	solveCorners(grid);

	while(solveGrid(grid) > .5){

	}

	printf("The average temperature is: %.2f\n", calculateGridAverage(grid));

	free(grid);



	/*
	* Cuda implementation.
	* currently non-functioning.
	*/

	int num_bytes = gridSize * gridSize * sizeof(float);

	float** grid_device;
	float** grid_destination;

	makeGrid(grid);
	solveCorners(grid);

	dim3 block_size(8,8);
	dim3 grid_size(1,1);

	size_t pitch;

	hipMallocPitch(&grid_device, &pitch, gridSize * sizeof(float), gridSize);
	kernel<<<grid_size,block_size>>>(grid_device);
	printf("kernel call%d\n", 1);
	printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(grid, grid_device, num_bytes, hipMemcpyDeviceToHost);
	printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
}
